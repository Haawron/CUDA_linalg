#include <hip/hip_runtime.h>

#include <iostream>
#include <iomanip>
using namespace std;


__global__ void func(int *a, int *b, int *c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int N = 100;
    cout << "hi" << endl;
    int *a = new int[N], *b = new int[N], *c = new int[N];  // alloc c?
    int *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N; i++) {
        a[i] = 2 * i;
        b[i] = -3 * i + 43;
    }
    hipMalloc((void**)&dev_a, sizeof(int) * N);
    hipMalloc((void**)&dev_b, sizeof(int) * N);
    hipMalloc((void**)&dev_c, sizeof(int) * N);
    hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
    func<<<(N + 15)/16, 16>>>(dev_a, dev_b, dev_c, N);
    hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        cout << setw(3) << c[i];
        if (i % 10 == 9) cout << endl;
    }
    cout << endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    delete[] a, b, c;
}